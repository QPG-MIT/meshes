
#include <hip/hip_runtime.h>
// meshes/gpu/gmem.cu
// Ryan Hamerly, 4/5/21
//
// Macros for interfacing with global memory:
//   load_u, load_u_du:   read from u_in [gmem] -> T [smem] -> u [regs]
//   save_u, save_u_du:   write from u [regs] -> T [smem] -> u_out [gmem]
//   load_T:              read from (s, p) [gmem] -> T [smem]
//   load_pos_cache:      read (lens, shifts) [gmem] -> (lens_cache, shifts_cache) [smem]
//
// History:
//   04/03/21: Implemented the algorithms in fwdprop.cu.
//   04/05/21: Generalized code with macros, split off to gmem.cu.
//   04/06/21: Added macros for back-propagation.
//   04/10/21: Added symmetric and orthogonal representations.


// Load u coalesced, gmem -> smem.  Then reorder strided, smem -> regs.  (Direct gmem -> regs would be strided, poor BW).
// Loads to smem can be parallelized over thread warps, but if threadIdx.y > 2*L, then there's not enough smem to do them
// all at once, hence the thread-sync'd for loop.
//
// It's a macro because I use it in multiple functions.  The code gets inlined anyway because the array u[] is in registers.
// Tried to write an inline function but multidimensional array arguments (of undefined size) seem unsupported.
//
// Python code that checks this (for deubugging):
/*
(B, L, K) = (4, 2, 2)    # Batch size, mesh length, mesh width/64.
ldu = 2*K*32; u_in = np.arange(B*ldu); T = np.zeros([L, 4*K, 32]).astype(int); u = np.zeros([B, 32, 2*K]).astype(int)

for i in range(0, B, 2*L):
    print (f"i = {i}")
    for tix_y in range(B):
        if (i <= tix_y < i + 2*L and tix_y < B):
            print (f"tix_y = {tix_y}")
            for k in range(2*K):
                ind = 32*k + ldu*tix_y
                print (f"idx=[{(tix_y//2)%L}][{k+2*K*(tix_y%2)}][:32] <-- [{ind}+:32]")
                for tix_x in range(32):
                    T[(tix_y//2)%L, k+2*K*(tix_y%2), tix_x] = u_in[ind+tix_x]
            for k in range(2*K):
                for tix_x in range(32):
                    u[tix_y, tix_x, k] = T[(tix_y//2)%L, (2*K*tix_x + k)//32 + 2*K*(tix_y%2), (2*K*tix_x + k)%32]       
(u.flatten() == u_in).all()
*/

// One-line code segments that get reused in the loaders below.
//
// load_readcode: reads u_in -> T.  
// load_shflcode: shuffles indices to load T into registers u[k].
// save_shflcode: shuffles indices to put registers u[k] into T.
// save_writcode: writes T -> u_out.
#define load_readcode(T, u_in, cond)    T[(l/2)%L0][k+2*K*(l%2)][threadIdx.x] = cond ? u_in[32*k + threadIdx.x] : 0
#define load_shflcode(T, u)             u[k] = T[(l/2)%L0][(2*K*threadIdx.x + k)/32+2*K*(l%2)][(2*K*threadIdx.x + k)%32]
#define save_shflcode(T, u)             T[(l/2)%L0][(2*K*threadIdx.x + k)/32+2*K*(l%2)][(2*K*threadIdx.x + k)%32] = u[k]
#define save_writcode(T, u_out, cond)   if (cond) {u_out[32*k + threadIdx.x] = T[(l/2)%L0][k+2*K*(l%2)][threadIdx.x];}

#define load_readcode_sym(T, u_in, cond) \
    {complex64 u_temp = cond ? u_in[32*k + threadIdx.x] : 0; \
    T[2*(l%(L0/2))+(k/K)][2*(k%K)  ][threadIdx.x] = u_temp.real(); \
    T[2*(l%(L0/2))+(k/K)][2*(k%K)+1][threadIdx.x] = u_temp.imag();}
#define save_writcode_sym(T, u_out, cond) \
    if (cond) {u_out[32*k + threadIdx.x] = \
        complex64(T[2*(l%(L0/2))+(k/K)][2*(k%K)  ][threadIdx.x], \
                  T[2*(l%(L0/2))+(k/K)][2*(k%K)+1][threadIdx.x]); }
#define load_shflcode_sym(T, u) \
    u[k] = complex64( \
              T[2*(l%(L0/2))+((2*K*threadIdx.x+k)/32/K)][2*(((2*K*threadIdx.x+k)/32)%K)  ][(2*K*threadIdx.x+k)%32], \
              T[2*(l%(L0/2))+((2*K*threadIdx.x+k)/32/K)][2*(((2*K*threadIdx.x+k)/32)%K)+1][(2*K*threadIdx.x+k)%32])
#define save_shflcode_sym(T, u) \
    T[2*(l%(L0/2))+((2*K*threadIdx.x+k)/32/K)][2*(((2*K*threadIdx.x+k)/32)%K)  ][(2*K*threadIdx.x+k)%32] = u[k].real(); \
    T[2*(l%(L0/2))+((2*K*threadIdx.x+k)/32/K)][2*(((2*K*threadIdx.x+k)/32)%K)+1][(2*K*threadIdx.x+k)%32] = u[k].imag();

#define load_readcode_orth(T, u_in, cond)    T[l%L0][k][threadIdx.x] = cond ? u_in[32*k + threadIdx.x] : 0
#define load_shflcode_orth(T, u)             u[k] = T[l%L0][(2*K*threadIdx.x + k)/32][(2*K*threadIdx.x + k)%32]
#define save_shflcode_orth(T, u)             T[l%L0][(2*K*threadIdx.x + k)/32][(2*K*threadIdx.x + k)%32] = u[k]
#define save_writcode_orth(T, u_out, cond)   if (cond) {u_out[32*k + threadIdx.x] = T[l%L0][k][threadIdx.x];}

// Generic loader.  Permutation keeps gmem reads coalesced.
// TODO -- combine these with a new variable L_inc = (2*L0, L0, or L0/2)
#define load_generic(L_inc, read_code, shuffle_code, zero_code) { \
for (int i = 0; i < b; i += L_inc) \
{ \
    int l = threadIdx.y - i; \
    if (0 <= l && l < L_inc && threadIdx.y < b) \
        for (int k = 0; k < 2*K; k++) \
            if (32*k + threadIdx.x < N) {read_code;} \
    __syncthreads(); \
    if (0 <= l && l < L_inc) \
        for (int k = 0; k < 2*K; k++) \
        { \
            if (2*K*threadIdx.x + k < N) {shuffle_code;} \
            else {zero_code;} \
        } \
    __syncthreads(); \
} \
}

// Generic saver.  Same permutation as for input, but reversed.
#define save_generic(L_inc, shuffle_code, save_code) { \
for (int i = 0; i < b; i += L_inc) \
{ \
    int l = threadIdx.y - i; \
    if (0 <= l && l < L_inc) \
        for (int k = 0; k < 2*K; k++) {shuffle_code;} \
    __syncthreads(); \
    if (0 <= l && l < L_inc && threadIdx.y < b) \
        for (int k = 0; k < 2*K; k++) \
            if (32*k + threadIdx.x < N) {save_code;} \
    __syncthreads(); \
} \
}

// Loads u_in -> u.
#define load_u_mzi(u, u_in) \
    load_generic(2*L0, load_readcode(T, u_in, true), load_shflcode(T, u), u[k] = 0)
#define load_u_sym(u, u_in) \
    load_generic(L0/2, load_readcode_sym(T, u_in, true), load_shflcode_sym(T, u), u[k] = 0)
#define load_u_orth(u, u_in) \
    load_generic(L0, load_readcode_orth(T, u_in, true), load_shflcode_orth(T, u), u[k] = 0)

// Loads u_in -> u, du_in -> du
#define load_u_du_mzi(u, du, u_in, du_in) \
    load_generic(2*L0, \
        load_readcode(T, u_in, true); load_readcode(dT, du_in, du_in), \
        load_shflcode(T, u); load_shflcode(dT, du), \
        u[k] = 0; du[k] = 0)
#define load_u_du_sym(u, du, u_in, du_in) \
    load_generic(L0/2, \
        load_readcode_sym(T, u_in, true); load_readcode_sym(dT, du_in, du_in), \
        load_shflcode_sym(T, u); load_shflcode_sym(dT, du), \
        u[k] = 0; du[k] = 0)
#define load_u_du_orth(u, du, u_in, du_in) \
    load_generic(L0, load_readcode_orth(T, u_in, true), load_shflcode_orth(T, u), u[k] = 0); \
    load_generic(L0, load_readcode_orth(T, du_in, true), load_shflcode_orth(T, du), du[k] = 0); \
// Saves u -> u_out.
#define save_u_mzi(u, u_out) \
    save_generic(2*L0, save_shflcode(T, u), save_writcode(T, u_out, true))
#define save_u_sym(u, u_out) \
    save_generic(L0/2, save_shflcode_sym(T, u), save_writcode_sym(T, u_out, true))
#define save_u_orth(u, u_out) \
    save_generic(L0, save_shflcode_orth(T, u), save_writcode_orth(T, u_out, true))
// Saves u -> u_out, du -> du_out.
#define save_u_du_mzi(u, du, u_out, du_out) \
    save_generic(2*L0, \
        save_shflcode(T, u); save_shflcode(dT, du), \
        save_writcode(T, u_out, true); save_writcode(dT, du_out, du_out))
#define save_u_du_sym(u, du, u_out, du_out) \
    save_generic(L0/2, \
        save_shflcode_sym(T, u); save_shflcode_sym(dT, du), \
        save_writcode_sym(T, u_out, true); save_writcode_sym(dT, du_out, du_out))
#define save_u_du_orth(u, du, u_out, du_out) \
    save_generic(L0, save_shflcode_orth(T, u), save_writcode_orth(T, u_out, true)); \
    save_generic(L0, save_shflcode_orth(T, du), save_writcode_orth(T, du_out, true)); \

// Every L_preload = L0*nL layers, reload the cache of shifts and lengths.  This is done less frequently than
// (p, s) updates because there's less data to load, so the cache can store more layers.  More importantly,
// the (p, s) updates rely on the shifts and lengths (i.e. to only load certain regions); doing these
// updates less frequently reduces memory latency.
/*
for (L0, nL, bd_y) in zip([36,20,14,11,7,5,4,2], [8,8,16,16,32,32,32,32], [8,10,15,18,26,20,16,12]):
    L = 1024; shifts = np.arange(L); shifts_cache = np.repeat(-1, [nL*L0]); shifts_cache_list = []
    for x in range(0, L, L0):
        if (x % (L0*nL) == 0):
            for i in range(0, L0*nL, 32*bd_y):
                for tix_y in range(bd_y):
                    for tix_x in range(32):
                        idx = i + 32*tix_y + tix_x
                        if (idx < L0*nL and x + idx < L):
                            shifts_cache[idx] = shifts[x + idx]
            shifts_cache_list.append(np.array(shifts_cache))
            shifts_cache[:] = -1
    print ((np.concatenate(shifts_cache_list)[:len(shifts)] == shifts).all())        
*/
#define load_cache(code) \
    if (x % L_preload == 0) \
    { \
        for (int i = 0; i < L_preload; i += 32*blockDim.y) \
        { \
            int id = i + 32*threadIdx.y + threadIdx.x; \
            if (id < L_preload && x + id < L) {code;} \
        } \
        __syncthreads(); \
    }

#define load_pos_cache(sign) \
    load_cache(lens_cache[id]   = lens[sign*(x + id)]; \
               shifts_cache[id] = shifts[sign*(x + id)])
#define load_pos_cache_fwd load_pos_cache(+1)
#define load_pos_cache_rev load_pos_cache(-1)

#define load_strides_cache(sign) \
    load_cache(strides_cache[id] = strides[sign*(x + id)])
#define load_strides_cache_fwd load_strides_cache(+1)
#define load_strides_cache_rev load_strides_cache(-1)




// Load T (coalesced in gmem, strided in smem).
// Python code that checks this (for debugging):
/*
B = 3    # Batch size (variable)
L = 10   # Length of mesh
K = 2    # Width of mesh: 64*K (2*K variables / thread)
ldp = lds = 2*K*32
p = np.array([1*np.arange(32*K*L), 2*np.arange(32*K*L)]).T.flatten()
s = np.array([3*np.arange(32*K*L), 4*np.arange(32*K*L)]).T.flatten()
T = np.zeros([L, 4*K, 32], dtype=int)

def T_test(p1, p2, s1, s2):
    return np.array([p1, p2, s1, s2])

for i in range(0, K*L, B):
    print (f"i={i}")
    for tix_y in range(B):
        print (f"   -> tix_y={tix_y}")
        l = (i + tix_y)//K; m = (i + tix_y)%K
        if (l < L):
            print (f"               -> m={m}, l={l}, idx={ldp*l + 2*32*m}")
            for tix_x in range(32):
                dm = (m*32 + tix_x)
                idx_p = ldp*l + 2*dm; idx_s = lds*l + 2*dm
                T[l, 4*(dm%K):4*(dm%K+1), dm//K] = T_test(p[idx_p], p[idx_p+1],
                                                          s[idx_s], s[idx_s+1])
(T.reshape(L*32*K, 4) == np.array([p[::2], p[1::2], s[::2], s[1::2]]).T).all()
*/

#define i1_T (l/1)
#define i2_T (stride_T*(dm%K + K*(l%1)))
#define i2_dth (stride_dth*(dm%K + K*(l%1)))
#define i3_T (dm/K)

#define IDX_PS(stride_p, stride_s)   int idx_p = ldp*l + stride_p*dm, idx_s = s ? (lds*l + stride_s*dm) : 0
#define IDX_P(stride_p)              int idx_p = ldp*l + stride_p*dm

#define cond_gen \
    dm >= shifts_cache[(x+l) % L_preload]/2 && \
    dm <  shifts_cache[(x+l) % L_preload]/2 + lens_cache[(x+l) % L_preload]
#define cond_fft \
    true

#define matrix_io(indexing, cond, code_in, code_out) { \
for (int i = 0; i < K*L_ker; i += blockDim.y) \
{ \
    int l = (i + threadIdx.y)/K, m = (i + threadIdx.y)%K; \
    if (l < L_blk) \
    { \
        int dm = (m*32 + threadIdx.x); \
        indexing; \
        if (cond) \
        { \
            code_in; \
        } \
        else \
        { \
            code_out; \
        } \
    } \
} \
__syncthreads(); \
}

// Loads matrix T.
#define ldT_m  Tij_mzi     (&p[idx_p], (float *) 0, &s[idx_s], &T[i1_T][i2_T][i3_T], (complex64 *) 0, false)
#define ldT_s  Tij_mzi_sym (&p[idx_p], (float *) 0, &s[idx_s], &T[i1_T][i2_T][i3_T], (float *) 0, mode, false)
#define ldT_o  Tij_mzi_orth(&p[idx_p], (float *) 0, &T[i1_T][i2_T][i3_T], (float *) 0, false)
#define idT_m  Tij_identity     (&T[i1_T][i2_T][i3_T], (complex64 *) 0)
#define idT_s  Tij_identity_sym (&T[i1_T][i2_T][i3_T], (float *) 0)
#define idT_o  Tij_identity_orth(&T[i1_T][i2_T][i3_T], (float *) 0)

#define load_T_mzi    matrix_io(IDX_PS(2, 2), cond_gen, ldT_m, idT_m)
#define load_T_sym    matrix_io(IDX_PS(2, 1), cond_gen, ldT_s, idT_s)
#define load_T_orth   matrix_io(IDX_P(1),     cond_gen, ldT_o, idT_o)
#define loadft_T_mzi  matrix_io(IDX_PS(2, 2), cond_fft, ldT_m,      )
#define loadft_T_sym  matrix_io(IDX_PS(2, 1), cond_fft, ldT_s,      )
#define loadft_T_orth matrix_io(IDX_P(1),     cond_fft, ldT_o,      )

// Loads matrix T and its differential dT.  For forward differentiation.
#define ldTf_m  Tij_mzi     (&p[idx_p], &dp[idx_p], &s[idx_s], &T[i1_T][i2_T][i3_T], &dT[i1_T][i2_T][i3_T], true)
#define ldTf_s  Tij_mzi_sym (&p[idx_p], &dp[idx_p], &s[idx_s], &T[i1_T][i2_T][i3_T], &dT[i1_T][i2_T][i3_T], mode, true)
#define ldTf_o  Tij_mzi_orth(&p[idx_p], &dp[idx_p], &T[i1_T][i2_T][i3_T], &dth[i1_T][i2_dth][i3_T], true)
#define idTf_m  Tij_identity     (&T[i1_T][i2_T][i3_T], &dT[i1_T][i2_T][i3_T])
#define idTf_s  Tij_identity_sym (&T[i1_T][i2_T][i3_T], &dT[i1_T][i2_T][i3_T])
#define idTf_o  Tij_identity_orth(&T[i1_T][i2_T][i3_T], &dth[i1_T][i2_dth][i3_T])

#define load_T_dT_mzi    matrix_io(IDX_PS(2, 2), cond_gen, ldTf_m, idTf_m)
#define load_T_dT_sym    matrix_io(IDX_PS(2, 1), cond_gen, ldTf_s, idTf_s)
#define load_T_dT_orth   matrix_io(IDX_P(1),     cond_gen, ldTf_o, idTf_o)
#define loadft_T_dT_mzi  matrix_io(IDX_PS(2, 2), cond_fft, ldTf_m,       )
#define loadft_T_dT_sym  matrix_io(IDX_PS(2, 1), cond_fft, ldTf_s,       )
#define loadft_T_dT_orth matrix_io(IDX_P(1),     cond_fft, ldTf_o,       )

// Loads matrix T and initializes dT = 0.  For back-propagation.
#define ldTb_m  Tij_mzi     (&p[idx_p], &dp[idx_p], &s[idx_s], &T[i1_T][i2_T][i3_T], &dT[i1_T][i2_T][i3_T], false)
#define ldTb_s  Tij_mzi_sym (&p[idx_p], &dp[idx_p], &s[idx_s], &T[i1_T][i2_T][i3_T], &dT[i1_T][i2_T][i3_T], mode, false)
#define ldTb_o  Tij_mzi_orth(&p[idx_p], &dp[idx_p], &T[i1_T][i2_T][i3_T], &dth[i1_T][i2_dth][i3_T], false)
#define idTb_m  Tij_identity     (&T[i1_T][i2_T][i3_T], &dT[i1_T][i2_T][i3_T])
#define idTb_s  Tij_identity_sym (&T[i1_T][i2_T][i3_T], &dT[i1_T][i2_T][i3_T])
#define idTb_o  Tij_identity_orth(&T[i1_T][i2_T][i3_T], &dth[i1_T][i2_dth][i3_T])

#define load_T_dT_bk_mzi    matrix_io(IDX_PS(2, 2), cond_gen, ldTb_m, idTb_m)
#define load_T_dT_bk_sym    matrix_io(IDX_PS(2, 1), cond_gen, ldTb_s, idTb_s)
#define load_T_dT_bk_orth   matrix_io(IDX_P(1),     cond_gen, ldTb_o, idTb_o)
#define loadft_T_dT_bk_mzi  matrix_io(IDX_PS(2, 2), cond_fft, ldTb_m,       )
#define loadft_T_dT_bk_sym  matrix_io(IDX_PS(2, 1), cond_fft, ldTb_s,       )
#define loadft_T_dT_bk_orth matrix_io(IDX_P(1),     cond_fft, ldTb_o,       )

// Save dp to global memory.  For back-propagation.
#define svdp_m  dp_mzi     (&p[idx_p], &s[idx_s], &dT[i1_T][i2_T][i3_T], &dp[idx_p])
#define svdp_s  dp_mzi_sym (&p[idx_p], &s[idx_s], &dT[i1_T][i2_T][i3_T], &dp[idx_p])
#define svdp_o  dp_mzi_orth(&dth[i1_T][i2_dth][i3_T], &dp[idx_p])

#define save_dp_mzi    matrix_io(IDX_PS(2, 2), cond_gen, svdp_m, )
#define save_dp_sym    matrix_io(IDX_PS(2, 1), cond_gen, svdp_s, )
#define save_dp_orth   matrix_io(IDX_P(1),     cond_gen, svdp_o, )
#define saveft_dp_mzi  matrix_io(IDX_PS(2, 2), cond_fft
#define saveft_dp_sym  matrix_io(IDX_PS(2, 1), cond_fft, svdp_s, )
#define saveft_dp_orth matrix_io(IDX_P(1),     cond_fft, svdp_o, )