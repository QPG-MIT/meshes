#include "hip/hip_runtime.h"
// meshes/gpu/backdiff.cu
// Ryan Hamerly, 4/6/21
//
// Device functions to handle the MZI arithmetic, namely obtaining (T, dT) from (p, dp, s) and back-propagating
// gradients of dT to the parameters p.
//
// History:
//   04/06/21: Moved this to its own file, added matmult_** routines and dp_mzi (for backprop).
//   04/10/21: Added symmetric and orthogonal representations.

// Initializes an identity transfer matrix [[1, 0], [0, 1]].
__device__ void Tij_identity(complex64 *T, complex64 *dT)
{
    T[0] = 1; T[32] = 0; T[64] = 0; T[96] = 1;
    if (dT)
    {
        dT[0] = 0; dT[32] = 0; dT[64] = 0; dT[96] = 0;
    }
}
__device__ void Tij_identity_sym(float *T, float *dT)
{
    T[0] = 1; T[32] = 0; T[64] = 0;
    if (dT)
    {
        dT[0] = 0; dT[32] = 0; dT[64] = 0;
    }
}
__device__ void Tij_identity_orth(float *T, float *dth)
{
    T[0] = 1; T[32] = 0;
    if (dth)
        dth[0] = 0;
}

// Initializes T = [T11, T12, T21, T22] to given MZI settings (θ, φ) and imperfections (α, β).
__device__ void Tij_mzi(const float *p, const float *dp, const float *s, complex64 *T, complex64 *dT, bool init_dT)
{
	// cos(θ/2), sin(θ/2), cos(θ/2+φ), sin(θ/2+φ)
	float C, S, C1, S1;
	__sincosf(0.5f*p[0],   &S , &C );
	__sincosf(p[0]/2+p[1], &S1, &C1);

	// cos(α ± β), sin(α ± β)
	float Cp, Sp, Cm, Sm;
	__sincosf(s[0]+s[1],   &Sp, &Cp);
	__sincosf(s[0]-s[1],   &Sm, &Cm);

	// Equivalent Python code:
    // (Cp, Cm, C, Sp, Sm, S) = [fn(x) for fn in [np.cos, np.sin] for x in [beta[0]+beta[1], beta[0]-beta[1], theta/2]]
    // T = np.exp(1j*theta/2) * np.array([[np.exp(1j*phi) * (1j*S*Cm - C*Sp),    1j*C*Cp - S*Sm],
    //                                    [np.exp(1j*phi) * (1j*C*Cp + S*Sm),   -1j*S*Cm - C*Sp]])
    T[0 ] = complex64(C1, S1) * complex64(-C*Sp,  S*Cm);
    T[32] = complex64(C , S ) * complex64(-S*Sm,  C*Cp);
    T[64] = complex64(C1, S1) * complex64( S*Sm,  C*Cp);
    T[96] = complex64(C , S ) * complex64(-C*Sp, -S*Cm);

    // Equivalent Python code:
    // dT = (np.exp(1j*np.array([[[phi+theta, theta]]])) *
    //       np.array([[[1j*(1j*S*Cm-C*Sp)+( 1j*C*Cm+S*Sp),   1j*( 1j*C*Cp-S*Sm)+(-1j*S*Cp-C*Sm)],
    //                  [1j*(1j*C*Cp+S*Sm)+(-1j*S*Cp+C*Sm),   1j*(-1j*S*Cm-C*Sp)+(-1j*C*Cm+S*Sp)]],
    //                 [[1j*(1j*S*Cm-C*Sp),                   0*S                               ],
    //                  [1j*(1j*C*Cp+S*Sm),                   0*S                               ]]]))
    if (dT)
    {
        if (init_dT)
        {
            dT[0 ] = complex64(C1, S1) * (0.5f * dp[0] * (Cm-Sp) * complex64(-S,  C) + dp[1] * complex64(-Cm*S, -C*Sp));
            dT[32] = complex64(C , S ) * (0.5f * dp[0] * (Cp+Sm) * complex64(-C, -S));
            dT[64] = complex64(C1, S1) * (0.5f * dp[0] * (Cp-Sm) * complex64(-C, -S) + dp[1] * complex64(-C*Cp,  S*Sm));
            dT[96] = complex64(C , S ) * (0.5f * dp[0] * (Cm+Sp) * complex64( S, -C));
        }
        else
        {
            dT[0] = 0; dT[32] = 0; dT[64] = 0; dT[96] = 0;
        }
    }
}

// Symmetric MZI design.
//
// T = [[exp(+i*φ) (sin(θ/2) + i cos(θ/2)sin(2α)),  i cos(θ/2)cos(2α)                       ],
//      [i cos(θ/2)cos(2α),                         exp(-i*φ) (sin(θ/2) - i cos(θ/2)sin(2α))]]
__device__ void Tij_mzi_sym(const float *p, const float *dp, const float *s, float *T, float *dT, int cartesian, bool init_dT)
{
    if (cartesian == 0)
    {
        // cos(θ/2), sin(θ/2), cos(φ), sin(φ)
        float C_th, S_th, C_ph, S_ph;
        __sincosf(0.5f*p[0], &S_th, &C_th);
        __sincosf(p[1],      &S_ph, &C_ph);

        // cos(2α), sin(2α), the imperfections (if any)
        float C_2a = 1, S_2a = 0;
        if (s != 0)
            __sincosf(2*s[0], &S_2a, &C_2a);

        T[0 ] = S_th*C_ph - C_th*S_ph*S_2a;
        T[32] = S_th*S_ph + C_th*C_ph*S_2a;
        T[64] = C_th*C_2a;

        if (dT)
        {
            if (init_dT)
            {
                dT[0 ] =  0.5f*dp[0]*(C_th*C_ph + S_th*S_ph*S_2a) - dp[1]*(S_th*S_ph + C_th*C_ph*S_2a);
                dT[32] =  0.5f*dp[0]*(C_th*S_ph - S_th*C_ph*S_2a) + dp[1]*(S_th*C_ph - C_th*S_ph*S_2a);
                dT[64] = -0.5f*dp[0]*S_th*C_2a;
            }
            else
            {
                dT[0] = 0; dT[32] = 0; dT[64] = 0;
            }
        }
    }
    else
    {
        // TODO -- handle case where cartesian=true        
    }
}

// Orthogonal MZI design.  Builds up SO(N), not U(N).
//
// T = [[sin(θ/2), -cos(θ/2)],
//      [cos(θ/2),  sin(θ/2)]]
__device__ void Tij_mzi_orth(const float *p, const float *dp, float *T, float *dth, bool init_dT)
{
	// cos(θ/2), sin(θ/2)
	float C, S;
	__sincosf(p[0]/2, &S, &C);
    
    T[0 ] =  S;
    T[32] = -C;

    if (dth)
    {
        if (init_dT)
            dth[0] = 0.5f*dp[0];
        else
            dth[0] = 0.0f;
    }
}

// Gets the gradients with respect to MZI parameters p = (θ, φ).  Only used in back-propagation.
// dJ/dp = dJ/dT_{ij} dT_{ij}/dp + c.c. = 2*Re[dJ/dT_{ij} dT_{ij}/dp]
__device__ void dp_mzi(const float *p, const float *s, const complex64 *dT, float *dp)
{
	// cos(θ/2), sin(θ/2), cos(θ/2+φ), sin(θ/2+φ)
	float C, S, C1, S1;
	__sincosf(p[0]/2,      &S , &C );
	__sincosf(p[0]/2+p[1], &S1, &C1);

	// cos(α ± β), sin(α ± β)
	float Cp, Sp, Cm, Sm;
	__sincosf(s[0]+s[1],   &Sp, &Cp);
	__sincosf(s[0]-s[1],   &Sm, &Cm);

    // TODO -- simplify this once I'm sure that it works.
    float dp0, dp1;
    dp0 = (dT[0 ] * complex64(C1, S1) * (0.5f * (Cm-Sp) * complex64(-S,  C)) + 
           dT[32] * complex64(C , S ) * (0.5f * (Cp+Sm) * complex64(-C, -S)) + 
           dT[64] * complex64(C1, S1) * (0.5f * (Cp-Sm) * complex64(-C, -S)) + 
           dT[96] * complex64(C , S ) * (0.5f * (Cm+Sp) * complex64( S, -C))).real();
    dp1 = (dT[0 ] * complex64(C1, S1) * (complex64(-Cm*S, -C*Sp)) + 
           dT[64] * complex64(C1, S1) * (complex64(-C*Cp,  S*Sm))).real();
    atomicAdd(&dp[0], dp0);
    atomicAdd(&dp[1], dp1);
}
__device__ void dp_mzi_sym(const float *p, const float *s, const float *dT, float *dp)
{
    // Initialize cos(...), sin(...).
	float C_th, S_th, C_ph, S_ph, C_2a = 1, S_2a = 0;
	__sincosf(p[0]/2, &S_th, &C_th);
	__sincosf(p[1],   &S_ph, &C_ph);
    if (s != 0)
        __sincosf(2*s[0], &S_2a, &C_2a);
    
    float dp0, dp1;
    dp0 = (dT[0 ] * (C_th*C_ph + S_th*S_ph*S_2a) + 
           -dT[32] * (C_th*S_ph - S_th*C_ph*S_2a) - 
           -dT[64] * S_th*C_2a) * 0.5f;
    dp1 = (-dT[0 ] * (S_th*S_ph + C_th*C_ph*S_2a) + 
            -dT[32] * (S_th*C_ph - C_th*S_ph*S_2a)) * 1.0f;
    atomicAdd(&dp[0], dp0);
    atomicAdd(&dp[1], dp1);
}
__device__ void dp_mzi_orth(const float *dth, float *dp)
{
    atomicAdd(&dp[0], 0.5f*dth[0]);
}

__device__ __inline__ void matmult(const complex64 *T, complex64 &u1, complex64 &u2, complex64 &temp, bool cond)
{
    // u_i -> T_{ij} u_j
    temp = T[0 ]*u1 + T[32]*u2;
    u2   = T[64]*u1 + T[96]*u2;
    if (cond)
        u1 = temp;
}
__device__ __inline__ void matmult_sym(const float *T, float &u1, float &v1, float &u2, float &v2, 
                                       float &temp1, float &temp2, float &temp3, bool cond)
{
    // u_i -> T_{ij} u_j
    temp1 =  T[0 ]*u1 - T[32]*v1 - T[64]*v2;
    temp2 =  T[32]*u1 + T[0 ]*v1 + T[64]*u2;
    temp3 =  T[0 ]*u2 + T[32]*v2 - T[64]*v1;
    v2    = -T[32]*u2 + T[0 ]*v2 + T[64]*u1;
    u2    =  temp3;
    if (cond)
    {
        u1 = temp1;
        v1 = temp2;
    }
}
__device__ __inline__ void matmult_orth(const float *T, float &u1, float &u2, float &temp, bool cond)
{
    temp =  T[0 ]*u1 + T[32]*u2;
    u2   = -T[32]*u1 + T[0 ]*u2;
    if (cond)
        u1 = temp;
}


__device__ __inline__ void matmult_d(const complex64 *T, const complex64 *dT, 
                                     complex64 &u1, complex64 &u2, complex64 &du1, complex64 &du2, 
                                     complex64 &temp, bool cond)
{
    // du_i -> T_{ij} du_j + dT_{ij} u_j
    temp = T[0 ]*du1 + T[32]*du2 + dT[0 ]*u1 + dT[32]*u2;
    du2  = T[64]*du1 + T[96]*du2 + dT[64]*u1 + dT[96]*u2;
    if (cond)
        du1 = temp;
    // u_i -> T_{ij} u_j
    matmult(T, u1, u2, temp, cond);
}
__device__ __inline__ void matmult_d_sym(const float *T, const float *dT, float &u1, float &v1, float &u2, float &v2, 
                                         float &du1, float &dv1, float &du2, float &dv2,
                                         float &temp1, float &temp2, float &temp3, bool cond)
{
    // du_i -> T_{ij} du_j + dT_{ij} u_j
    temp1 =  T[0 ]*du1 - T[32]*dv1 - T[64]*dv2 + dT[0 ]*u1 - dT[32]*v1 - dT[64]*v2;
    temp2 =  T[32]*du1 + T[0 ]*dv1 + T[64]*du2 + dT[32]*u1 + dT[0 ]*v1 + dT[64]*u2; 
    temp3 =  T[0 ]*du2 + T[32]*dv2 - T[64]*dv1 + dT[0 ]*u2 + dT[32]*v2 - dT[64]*v1;
    dv2   = -T[32]*du2 + T[0 ]*dv2 + T[64]*du1 - dT[32]*u2 + dT[0 ]*v2 + dT[64]*u1;
    du2   =  temp3;
    if (cond)
    {
        du1 = temp1;
        dv1 = temp2;
    }
    // u_i -> T_{ij} u_j
    matmult_sym(T, u1, v1, u2, v2, temp1, temp2, temp3, cond);
}
__device__ __inline__ void matmult_d_orth(const float *T, const float *dth, float &u1, float &u2, 
                                          float &du1, float &du2, float &temp, bool cond)
{
    temp =  T[0 ]*du1 + T[32]*du2 + dth[0]*(-T[32]*u1 + T[0 ]*u2);
    du2  = -T[32]*du1 + T[0 ]*du2 + dth[0]*(-T[ 0]*u1 - T[32]*u2);
    if (cond)
        du1 = temp;
    matmult_orth(T, u1, u2, temp, cond);
}


__device__ __inline__ void atomicAdd(complex64 *A, complex64 B)
{
    float *A_float = (float *) A;
    atomicAdd(A_float,   B.real());
    atomicAdd(A_float+1, B.imag());
}

// Back-propagation of signals and gradients.
// Here, (dJdu1, dJdu2) represent the gradients dJ/du*, which is conjugate to dJ/du.
// TODO: check that conj(A)*B is properly compiled (takes as many FLOPS as A*B).  Otherwise, pre-conjugate T.
__device__ __inline__ void matmult_bk(const complex64 *T, complex64 *dT, 
                                      complex64 &u1, complex64 &u2, complex64 &dJdu1, complex64 &dJdu2, 
                                      complex64 &temp, bool cond)
{
    // u_i -> (T^dag)_{ij} u_j = (T_{ji})^* u_j
    temp = conj(T[0 ])*u1 + conj(T[64])*u2;
    u2   = conj(T[32])*u1 + conj(T[96])*u2;
    if (cond)
        u1 = temp;
    // dJ/dT_{ij} = (dJ/du_i)_{out} (u_j)_{in} = (dJ/du_i^*)_{out}^* (u_j)_{in} 
    atomicAdd(&dT[0 ], conj(dJdu1)*u1); atomicAdd(&dT[32], conj(dJdu1)*u2); 
    atomicAdd(&dT[64], conj(dJdu2)*u1); atomicAdd(&dT[96], conj(dJdu2)*u2); 
    // dJ/du_i^* -> (T^*)_{ij} dJ/du_j = (T_{ji}) dJ/du_j^*
    temp  = conj(T[0 ])*dJdu1 + conj(T[64])*dJdu2;
    dJdu2 = conj(T[32])*dJdu1 + conj(T[96])*dJdu2;
    if (cond)
        dJdu1 = temp;
}
__device__ __inline__ void matmult_bk_sym(const float *T, float *dT, float &u1, float &v1, float &u2, float &v2, 
                                          float &dJdu1, float &dJdv1, float &dJdu2, float &dJdv2, 
                                          float &temp1, float &temp2, float &temp3, bool cond)
{
    // u -> (T^dag) u
    temp1 =  T[0 ]*u1 + T[32]*v1 + T[64]*v2;
    temp2 = -T[32]*u1 + T[0 ]*v1 - T[64]*u2;
    temp3 =  T[0 ]*u2 - T[32]*v2 + T[64]*v1;
    v2    =  T[32]*u2 + T[0 ]*v2 - T[64]*u1;
    u2    =  temp3;
    if (cond)
    {
        u1 = temp1;
        v1 = temp2;
    }
    // dJ/dT = (dJ/du*)* u^T
    atomicAdd(&dT[0 ],  u1*dJdu1 + v1*dJdv1 + u2*dJdu2 + v2*dJdv2);
    atomicAdd(&dT[32], -u1*dJdv1 + v1*dJdu1 + u2*dJdv2 - v2*dJdu2);
    atomicAdd(&dT[64], -u1*dJdv2 + v1*dJdu2 - u2*dJdv1 + v2*dJdu1);
    // dJ/du* -> (T^dag) dJ/du*
    temp1 =  T[0 ]*dJdu1 + T[32]*dJdv1 + T[64]*dJdv2;
    temp2 = -T[32]*dJdu1 + T[0 ]*dJdv1 - T[64]*dJdu2;
    temp3 =  T[0 ]*dJdu2 - T[32]*dJdv2 + T[64]*dJdv1;
    dJdv2 =  T[32]*dJdu2 + T[0 ]*dJdv2 - T[64]*dJdu1;
    dJdu2 =  temp3;
    if (cond)
    {
        dJdu1 = temp1;
        dJdv1 = temp2;
    }
}
__device__ __inline__ void matmult_bk_orth(const float *T, float *d_th, float &u1, float &u2,
                                           float &dJdu1, float &dJdu2, float &temp, bool cond)
{
    temp = T[0 ]*u1 - T[32]*u2;
    u2   = T[32]*u1 + T[0 ]*u2;
    if (cond)
        u1 = temp;
    atomicAdd(d_th, T[0]*(u2*dJdu1 - u1*dJdu2) - T[32]*(u1*dJdu1 + u2*dJdu2));
    temp  = T[0 ]*dJdu1 - T[32]*dJdu2;
    dJdu2 = T[32]*dJdu1 + T[0 ]*dJdu2;
    if (cond)
        dJdu1 = temp;
}