#include "hip/hip_runtime.h"
// meshes/gpu/backdiff.cu
// Ryan Hamerly, 4/6/21
//
// Implements the back-propagation function with differentiation backdiff_N[64*K](), where [64*K] is the mesh size.  
// Requires the following preprocessor directives:
//   K  [int] = size/32.  Each thread manages 2*K waveguides.
//   L0 [int] = number of layers natively supported.  Limited by smem.  If L > L0, the propagation is broken into steps.
//   nL [int] = a total of nL*L0 shifts/lens are pre-loaded.  Must be even.  Tradeoff between smem space and gmem latency.
//   fname    = name of function (should be backdiff_N[64*K])
//
// History:
//   04/06/21: Created this file.  First working code with back-propagation.
//   05/17/21: Shortened and simplified, merging the 3 crossing types.


#define L_ker (L0)  // Actual number of layers stored in the kernel = L0*pack_T (default L0, sym: 2*L0).
#define L_preload (L0*nL)  // Number of shifts / lens pre-loaded.

#if   CROSSING_TYPE == MZI
    #define stride_T    4
    #define stride_dT   4
    #define define_T    __shared__ complex64 T[L0][4*K][32], dT[L0][4*K][32]
    #define load_u_du   load_u_du_mzi(u, dJdu, u_out, dJdu_out)
    #define load_T_dT   load_T_dT_bk_mzi
    #define save_u_du   save_u_du_mzi(u, dJdu, u_in,  dJdu_in)
    #define save_dp     save_dp_mzi
    #define matmult_bk  matmult_bk_mzi
    #define scalar      complex64
#elif CROSSING_TYPE == SYM
    #define stride_T    3
    #define stride_dT   3
    #define define_T    __shared__ float T[L0][3*K][32], dT[L0][3*K][32]
    #define load_u_du   load_u_du_sym(u, dJdu, u_out, dJdu_out)
    #define load_T_dT   load_T_dT_bk_sym
    #define save_u_du   save_u_du_sym(u, dJdu, u_in,  dJdu_in)
    #define save_dp     save_dp_sym
    #define matmult_bk  matmult_bk_sym
    #define scalar      complex64
#else
    #define stride_T    2
    #define stride_dT   1
    #define dth         dT
    #define stride_dth  stride_dT
    #define define_T    __shared__ float T[L0][2*K][32], dth[L0][K][32]
    #define load_u_du   load_u_du_orth(u, dJdu, u_out, dJdu_out)
    #define load_T_dT   load_T_dT_bk_orth
    #define save_u_du   save_u_du_orth(u, dJdu, u_in,  dJdu_in)
    #define save_dp     save_dp_orth
    #define matmult_bk  matmult_bk_orth
    #define scalar      float
#endif


__global__ void fname(int N, int L, int B, int *lens, int *shifts, 
                      float *p, float *dp, int ldp, float *s, int lds, 
                      scalar *u_out, scalar *dJdu_out,
                      scalar *u_in,  scalar *dJdu_in,  int ldu, int mode)
{
    // Definitions and Initializations.
	u_out    += ldu * (blockDim.y*blockIdx.x + threadIdx.y);    // Pointer shift, one warp per instance.
	u_in     += ldu * (blockDim.y*blockIdx.x + threadIdx.y);
    dJdu_out += ldu * (blockDim.y*blockIdx.x + threadIdx.y);
    if (dJdu_in) {dJdu_in += ldu * (blockDim.y*blockIdx.x + threadIdx.y);}
    int b = (blockDim.y*(1 + blockIdx.x) < B) ? (blockDim.y) : (B - blockDim.y*blockIdx.x);     // # active warps
    p  += ldp*(L-1);  lens   += (L-1);                          // Flip arrays since we're iterating backwards.
    dp += ldp*(L-1);  shifts += (L-1);  
    if (s) {s += lds*(L-1);}
    ldp *= -1; lds *= -1;
    define_T;                                                   // Transfer matrices T, dT (dim=[L0][s*K][32]).
    __shared__ int shifts_cache[L0*nL], lens_cache[L0*nL];      // Cache of lengths, shifts
	scalar u[2*K], dJdu[2*K];                                   // State and gradient.
    load_u_du;                                                  // Load u and dJdu, gmem -> smem [macro: gmem.cu].

    // Propagate fields through the mesh.
	for (int x = 0; x < L; x += L_ker)
    {
        int L_blk = (L_ker < L-x) ? L_ker : L-x;                // Layers in block = min(L0, L-x)
        load_pos_cache_rev;                                     // Occasionally reload cache of shifts / lengths.
        load_T_dT;                                              // Load transfer matrices [macro: gmem.cu].

        // Iterate through L_blk layers.
        if (threadIdx.y < b)
        {
            for (int l = 0; l < L_blk; l++)                     // Iterate through L_blk layers.
            {
                scalar temp, u_2k, dJdu_2k;
                if (shifts_cache[(x+l) % L_preload] % 2)        // Misaligned MZIs: need warp shuffle.
                {
                    for (int i = 0; i < K-1; i++)               // Couple (u[1], u[2]), ..., (u[2K-3], u[2K-2])
                        matmult_bk(&T[l][stride_T*i][threadIdx.x], &dT[l][stride_dT*i][threadIdx.x], 
                                   u[2*i+1], u[2*i+2], dJdu[2*i+1], dJdu[2*i+2], temp, true);
                    // Couple (u[2K-1], u[0]) with warp shuffle.  
                    u_2k = __shfl_down_sync(0xffffffffu, u[0], 1, 32); dJdu_2k = __shfl_down_sync(0xffffffffu, dJdu[0], 1, 32);
                    matmult_bk(&T[l][stride_T*K-stride_T][threadIdx.x], &dT[l][stride_dT*K-stride_dT][threadIdx.x], 
                               u[2*K-1], u_2k, dJdu[2*K-1], dJdu_2k, temp, threadIdx.x != 31);
                    u_2k = __shfl_up_sync(0xffffffffu, u_2k, 1, 32); dJdu_2k = __shfl_up_sync(0xffffffffu, dJdu_2k, 1, 32);
                    if (threadIdx.x)
                    {
                        u[0]  = u_2k;
                        dJdu[0] = dJdu_2k;
                    }
                }
                else                                            // Aligned MZIs.  Easy case!
                {
                    for (int i = 0; i < K; i++)                 // Couple (u[0], u[1]), ... (u[2K-2], u[2K-1]).
                        matmult_bk(&T[l][stride_T*i][threadIdx.x], &dT[l][stride_dT*i][threadIdx.x], 
                                       u[2*i], u[2*i+1], dJdu[2*i], dJdu[2*i+1], temp, true);
                }
            }
        }
        
        __syncthreads();
        save_dp;
        p  += L_ker * ldp;
        dp += L_ker * ldp;
        if (s) {s += L_ker * lds;}
        __syncthreads();
    }

	// Write data to output.  Same permutation as for input, but reversed.  Macro from meshprop.cu.
    save_u_du;
}


#undef L_ker
#undef L_preload
#undef K
#undef L0
#undef nL
#undef fname
#undef stride_T
#undef stride_dT
#undef define_T
#undef load_u_du
#undef load_T_dT
#undef save_u_du
#undef save_dp
#undef matmult_bk
#undef scalar
#if CROSSING_TYPE == ORTH
    #undef dth
    #undef stride_dth
#endif