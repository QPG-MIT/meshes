#include "hip/hip_runtime.h"
// meshes/gpu/backdiff.cu
// Ryan Hamerly, 4/6/21
//
// Implements the back-propagation function with differentiation backdiff_N[64*K](), where [64*K] is the mesh size.  
// Requires the following preprocessor directives:
//   K  [int] = size/32.  Each thread manages 2*K waveguides.
//   L0 [int] = number of layers natively supported.  Limited by smem.  If L > L0, the propagation is broken into steps.
//   nL [int] = a total of nL*L0 shifts/lens are pre-loaded.  Must be even.  Tradeoff between smem space and gmem latency.
//   fname    = name of function (should be backdiff_N[64*K])
//
// History:
//   04/06/21: Created this file.  First working code with back-propagation.


#define L_ker (L0*pack_T)  // Actual number of layers stored in the kernel = L0*pack_T (default L0, sym: 2*L0).
#define L_preload (L0*nL)  // Number of shifts / lens pre-loaded.

__global__ void fname(int N, int L, int B, 
                      int *lens, int *shifts, 
                      float *p, float *dp, int ldp, 
                      float *s, int lds, 
                      complex64 *u_out, complex64 *dJdu_out,
                      complex64 *u_in,  complex64 *dJdu_in,  int ldu)
{
    const int pack_u = 2; // Packing factor = T.shape[2]/2 (default 2)
    const int pack_T = 1; // Packing factor 4 / (# T params) (default: 1, symmetric Tij: 2)
    const int stride_T = 4 / pack_T;

    // There are blockDim.y warps in each block (blockDim.x = 32).  Each references a separate instance.
	// The blocks are therefore offset by blockDim.y instances, i.e. a pointer offset of ld * blockDim.y
	// Kernel doesn't support multiplexing over p, s.  This is assumed to be easier by calling separate kernels.
	u_out     += ldu * (blockDim.y*blockIdx.x + threadIdx.y);
	u_in      += ldu * (blockDim.y*blockIdx.x + threadIdx.y);
    dJdu_out  += ldu * (blockDim.y*blockIdx.x + threadIdx.y);
    if (dJdu_in) {dJdu_in += ldu  * (blockDim.y*blockIdx.x + threadIdx.y);}
    
    // Number of active warps (this block's mini-batch).
    int b = (blockDim.y*(1 + blockIdx.x) < B) ? (blockDim.y) : (B - blockDim.y*blockIdx.x);
    
    // Since we're iterating backwards through the mesh, need to move the (s, p, dp) pointers to the final layer and
    // flip the signs of (lds, ldp).
    p      += ldp * (L-1);
    dp     += ldp * (L-1);
    s      += lds * (L-1);
    lens   += (L-1);
    shifts += (L-1);
    ldp    *= -1;
    lds    *= -1;
		
	// Transfer matrices.
	// The b^th matrix of column c goes in T[c][4(b%K):4(b%K)+4][b/K].
	__shared__ complex64 T[L0][4*K][32];
	__shared__ complex64 dT[L0][4*K][32];
    __shared__ int shifts_cache[L_preload];
    __shared__ int lens_cache[L_preload];
    
	// State.  The i^th waveguide is u[i%K] of thread i/K.
	complex64 u[2*K];
	complex64 dJdu[2*K];
	
	// Load u coalesced, gmem -> smem.  Macro defined in meshprop.cu.
    load_u_du(u, dJdu, u_out, dJdu_out);

	for (int x = 0; x < L; x += L_ker)
    {
        // Number of layers in *this* block.  Normally L0, except if last block is truncated.
        int L_blk = (L_ker < L-x) ? L_ker : L-x;

        // Preload shifts and lengths to the cache (macro from gmem.cu).
        load_pos_cache_rev;

        // Load T (coalesced in gmem, strided in smem).  
        load_T_dT_bk;

        // Iterate through L_blk layers.
        if (threadIdx.y < b)
        {
            for (int l = 0; l < L_blk; l++)
            {
                complex64 temp, u_2k, dJdu_2k;
                if (shifts_cache[(x+l) % L_preload] % 2) //((x+l) % 2)
                {
                    // Couple (u[1], u[2]), (u[3], u[4]), ... (u[2K-3], u[2K-2]).
                    for (int i = 0; i < K-1; i++)
                        matmult_bk(&T[l][4*i][threadIdx.x], &dT[l][4*i][threadIdx.x], 
                                   u[2*i+1], u[2*i+2], dJdu[2*i+1], dJdu[2*i+2], temp, true);
                    // Couple (u[2K-1], u[0]).  The latter comes from the next thread up.  Warp shuffle.
                    u_2k = __shfl_down_sync(0xffffffffu, u[0], 1, 32); dJdu_2k = __shfl_down_sync(0xffffffffu, dJdu[0], 1, 32);
                    matmult_bk(&T[l][4*K-4][threadIdx.x], &dT[l][4*K-4][threadIdx.x], 
                               u[2*K-1], u_2k, dJdu[2*K-1], dJdu_2k, temp, threadIdx.x != 31);
                    u_2k = __shfl_up_sync(0xffffffffu, u_2k, 1, 32); dJdu_2k = __shfl_up_sync(0xffffffffu, dJdu_2k, 1, 32);
                    if (threadIdx.x)
                    {
                        u[0]  = u_2k;
                        dJdu[0] = dJdu_2k;
                    }
                }
                else
                {
                    // Easy case!  Couple (u[0], u[1]), (u[2], u[3]), ... (u[2K-2], u[2K-1]).
                    for (int i = 0; i < K; i++)
                        matmult_bk(&T[l][4*i][threadIdx.x], &dT[l][4*i][threadIdx.x], 
                                   u[2*i], u[2*i+1], dJdu[2*i], dJdu[2*i+1], temp, true);
                }
            }
        }
        
        __syncthreads();
        
        save_dp;
        
        p  += L_ker * ldp;
        dp += L_ker * ldp;
        s  += L_ker * lds;
        
        __syncthreads();  // TODO -- is this necessary?
    }

	// Write data to output.  Same permutation as for input, but reversed.  Macro from meshprop.cu.
    save_u_du(u, dJdu, u_in, dJdu_in);
}

#undef L_ker
#undef L_preload
#undef K
#undef L0
#undef nL
#undef fname