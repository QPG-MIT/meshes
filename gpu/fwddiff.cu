#include "hip/hip_runtime.h"
// meshes/gpu/fwddiff.cu
// Ryan Hamerly, 4/5/21
//
// Implements the foward-propagation function with differentiation fwddiff_N[64*K](), where [64*K] is the mesh size.  
// Requires the following preprocessor directives:
//   K  [int] = size/32.  Each thread manages 2*K waveguides.
//   L0 [int] = number of layers natively supported.  Limited by smem.  If L > L0, the propagation is broken into steps.
//   nL [int] = a total of nL*L0 shifts/lens are pre-loaded.  Tradeoff between smem space and gmem latency.
//   fname    = name of function (should be fwdprop_N[64*K])
//
// History:
//   04/05/21: First working CUDA code.

#ifdef is_matmult_d
#else
__device__ __inline__ void matmult_d(const complex64 T[4], const complex64 dT[4], 
                                     complex64 &u1, complex64 &u2, complex64 &du1, complex64 &du2, 
                                     complex64 &temp, bool cond)
{
    temp = T[0]*du1 + T[1]*du2 + dT[0]*u1 + dT[1]*u2;  // <-- TODO -- first check uncoupled case.
    du2  = T[2]*du1 + T[3]*du2 + dT[2]*u1 + dT[3]*u2;
    if (cond)
        du1 = temp;
    temp = T[0]*u1 + T[1]*u2;
    u2   = T[2]*u1 + T[3]*u2;
    if (cond)
        u1 = temp;
}
#define is_matmult_d 1
#endif

#define L_ker (L0*pack_T)  // Actual number of layers stored in the kernel = L0*pack_T (default L0, sym: 2*L0).
#define L_preload (L0*nL)  // Number of shifts / lens pre-loaded.

__global__ void fname(int N, int L, int B, 
                      int *lens, int *shifts, 
                      float *p, float *dp, int ldp, 
                      float *s, int lds, 
                      complex64 *u_in, complex64 *du_in, int ld_in,
                      complex64 *u_out, complex64 *du_out, int ld_out)
{
    const int pack_u = 2; // Packing factor = T.shape[2]/2 (default 2)
    const int pack_T = 1; // Packing factor 4 / (# T params) (default: 1, symmetric Tij: 2)
    const int stride_T = 4 / pack_T;

    // There are blockDim.y warps in each block (blockDim.x = 32).  Each references a separate instance.
	// The blocks are therefore offset by blockDim.y instances, i.e. a pointer offset of ld * blockDim.y
	// Kernel doesn't support multiplexing over p, s.  This is assumed to be easier by calling separate kernels.
	u_in  += ld_in  * (blockDim.y*blockIdx.x + threadIdx.y);
	u_out += ld_out * (blockDim.y*blockIdx.x + threadIdx.y);
    if (du_in)  {du_in  += ld_in  * (blockDim.y*blockIdx.x + threadIdx.y);}
    if (du_out) {du_out += ld_out * (blockDim.y*blockIdx.x + threadIdx.y);}
    
    // Number of active warps (this block's mini-batch).
    int b = (blockDim.y*(1 + blockIdx.x) < B) ? (blockDim.y) : (B - blockDim.y*blockIdx.x);
    
		
	// Transfer matrices.
	// The b^th matrix of column c goes in T[c][b/K][4(b%K):4(b%K)+4].  TODO: Offset to avoid bank conflicts?
	__shared__ complex64 T[L0][32][4*K+1];
	__shared__ complex64 dT[L0][32][4*K+1];
    __shared__ int shifts_cache[nL*L0];
    __shared__ int lens_cache[nL*L0];
    
	// State.  The i^th waveguide is u[i%K] of thread i/K.
	complex64 u[2*K];
	complex64 du[2*K];
	
	// Load u coalesced, gmem -> smem.  Macro defined in meshprop.cu.
    load_u_du;

	for (int x = 0; x < L; x += L_ker)
    {
        // Number of layers in *this* block.  Normally L0, except if last block is truncated.
        int L_blk = (L_ker < L-x) ? L_ker : L-x;

        // Every L0*nL layers, reload the cache of shifts and lengths.  This is done less frequently than the
        load_pos_cache;

        // Load T (coalesced in gmem, strided in smem).  
        load_T(Tij_mzi(&p[idx_p], &dp[idx_p], &s[idx_s], 
                       &T[l/pack_T][dm/K][stride_T*(dm%K + K*(l%pack_T))], 
                       &dT[l/pack_T][dm/K][stride_T*(dm%K + K*(l%pack_T))]),
               Tij_identity(&T[l][dm/K][4*(dm%K)], 
                            &dT[l][dm/K][4*(dm%K)]));

        // Iterate through L_blk layers.
        for (int l = 0; l < L_blk; l++)
        {
            complex64 temp, u_2k, du_2k;
            if (shifts_cache[(x+l) % L_preload] % 2) //((x+l) % 2)
            {
                // Couple (u[1], u[2]), (u[3], u[4]), ... (u[2K-3], u[2K-2]).
                for (int i = 0; i < K-1; i++)
                    matmult_d(&T[l][threadIdx.x][4*i], &dT[l][threadIdx.x][4*i], 
                              u[2*i+1], u[2*i+2], du[2*i+1], du[2*i+2], temp, true);
                // Couple (u[2K-1], u[0]).  The latter comes from the next thread up.  Warp shuffle.
                u_2k = __shfl_down_sync(0xffffffffu, u[0], 1, 32); du_2k = __shfl_down_sync(0xffffffffu, du[0], 1, 32);
                matmult_d(&T[l][threadIdx.x][4*K-4], &dT[l][threadIdx.x][4*K-4], 
                          u[2*K-1], u_2k, du[2*K-1], du_2k, temp, threadIdx.x != 31);
                u_2k = __shfl_up_sync(0xffffffffu, u_2k, 1, 32); du_2k = __shfl_up_sync(0xffffffffu, du_2k, 1, 32);
                if (threadIdx.x)
                {
                    u[0]  = u_2k;
                    du[0] = du_2k;
                }
            }
            else
            {
                // Easy case!  Couple (u[0], u[1]), (u[2], u[3]), ... (u[2K-2], u[2K-1]).
                for (int i = 0; i < K; i++)
                    matmult_d(&T[l][threadIdx.x][4*i], &dT[l][threadIdx.x][4*i], 
                              u[2*i], u[2*i+1], du[2*i], du[2*i+1], temp, true);
            }
        }
        
        p  += L_ker * ldp;
        dp += L_ker * ldp;
        s  += L_ker * lds;
        
        __syncthreads();  // TODO -- is this necessary?
    }

	// Write data to output.  Same permutation as for input, but reversed.  Macro from meshprop.cu.
    save_u_du;
}

#undef L_ker
#undef L_preload
#undef K
#undef L0
#undef nL
#undef fname