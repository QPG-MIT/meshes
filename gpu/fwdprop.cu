#include "hip/hip_runtime.h"
// meshes/gpu/fwdprop.cu
// Ryan Hamerly, 4/3/21
//
// Implements the foward-propagation function fwdprop_N[64*K](), where [64*K] is the mesh size.  Requires the following
// preprocessor directives:
//   K  [int] = size/32.  Each thread manages 2*K waveguides.
//   L0 [int] = number of layers natively supported.  Limited by smem.  If L > L0, the propagation is broken into steps.
//   nL [int] = a total of nL*L0 shifts/lens are pre-loaded.  Must be even.  Tradeoff between smem space and gmem latency.
//   fname    = name of function (should be fwdprop_N[64*K])
//
// History:
//   04/03/21: Created this file.  First working CUDA code.
//   04/05/21: Moved the global memory I/O stuff to its own macros in gmem.cu.


#define L_ker (L0*pack_T)  // Actual number of layers stored in the kernel = L0*pack_T (default L0, sym: 2*L0).
#define L_preload (L0*nL)  // Number of shifts / lens pre-loaded.

#if CROSSING_TYPE == MZI
__global__ void fname(int N, int L, int B, 
                      int *lens, int *shifts, 
                      float *p, int ldp, 
                      float *s, int lds, 
                      complex64 *u_in,
                      complex64 *u_out, int ldu)
{
    const int pack_u = 2; // Packing factor = T.shape[2]/2 (default 2)
    const int pack_T = 1; // Packing factor 4 / (# T params) (default: 1, symmetric Tij: 2)
    const int stride_T = 4 / pack_T;
    const int stride_s = 2;

    // There are blockDim.y warps in each block (blockDim.x = 32).  Each references a separate instance.
	// The blocks are therefore offset by blockDim.y instances, i.e. a pointer offset of ld * blockDim.y
	// Kernel doesn't support multiplexing over p, s.  This is assumed to be easier by calling separate kernels.
	u_in  += ldu * (blockDim.y*blockIdx.x + threadIdx.y);
	u_out += ldu * (blockDim.y*blockIdx.x + threadIdx.y);
    // Number of active warps (this block's mini-batch).
    int b = (blockDim.y*(1 + blockIdx.x) < B) ? (blockDim.y) : (B - blockDim.y*blockIdx.x);
    
	// Transfer matrices.
	// The b^th matrix of column c goes in T[c][4(b%K):4(b%K)+4][b/K].
	__shared__ complex64 T[L0][4*K][32];
    __shared__ int shifts_cache[nL*L0];
    __shared__ int lens_cache[nL*L0];
    
	// State.  The i^th waveguide is u[i%K] of thread i/K.
	complex64 u[2*K];
	
	// Load u coalesced, gmem -> smem.  Macro defined in meshprop.cu.
    load_u(u, u_in);

	for (int x = 0; x < L; x += L_ker)
    {
        // Number of layers in *this* block.  Normally L0, except if last block is truncated.
        int L_blk = (L_ker < L-x) ? L_ker : L-x;

        // Every L0*nL layers, reload the cache of shifts and lengths.  This is done less frequently than the
        load_pos_cache_fwd;

        // Load T (coalesced in gmem, strided in smem). 
        load_T;

        // Iterate through L_blk layers.
        for (int l = 0; l < L_blk; l++)
        {
            complex64 temp, u_2k;
            if (shifts_cache[(x+l) % L_preload] % 2)
            {
                // Couple (u[1], u[2]), (u[3], u[4]), ... (u[2K-3], u[2K-2]).
                for (int i = 0; i < K-1; i++)
                    matmult(&T[l][4*i][threadIdx.x], u[2*i+1], u[2*i+2], temp, true);
                // Couple (u[2K-1], u[0]).  The latter comes from the next thread up.  Warp shuffle.
                u_2k = __shfl_down_sync(0xffffffffu, u[0], 1, 32);
                matmult(&T[l][4*K-4][threadIdx.x], u[2*K-1], u_2k, temp, threadIdx.x != 31);
                u_2k = __shfl_up_sync(0xffffffffu, u_2k, 1, 32);
                if (threadIdx.x)
                    u[0] = u_2k;
            }
            else
            {
                // Easy case!  Couple (u[0], u[1]), (u[2], u[3]), ... (u[2K-2], u[2K-1]).
                for (int i = 0; i < K; i++)
                    matmult(&T[l][4*i][threadIdx.x], u[2*i], u[2*i+1], temp, true);
            }
        }
        
        p += L_ker * ldp;
        s += L_ker * lds;
        
        __syncthreads();  // TODO -- is this necessary?
    }

	// Write data to output.  Same permutation as for input, but reversed.  Macro from meshprop.cu.
    save_u(u, u_out);
}
#endif



#if CROSSING_TYPE == SYM
__global__ void fname(int N, int L, int B, int *lens, int *shifts, float *p, int ldp, float *s, int lds, 
                      complex64 *u_in, complex64 *u_out, int ldu, bool cartesian)
{
    const int pack_T = 1, stride_T = 3, stride_s = 1;

	u_in  += ldu * (blockDim.y*blockIdx.x + threadIdx.y); 
    u_out += ldu * (blockDim.y*blockIdx.x + threadIdx.y);
    int b = (blockDim.y*(1 + blockIdx.x) < B) ? (blockDim.y) : (B - blockDim.y*blockIdx.x);   // # active warps
    __shared__ int shifts_cache[L_preload], lens_cache[L_preload];
	__shared__ float T[L0][3*K][32];   // Transfer matrix.  T = [[a+i*b, i*c], [i*c, a-i*b]], store floats (a, b, c).
	float u[2*K], v[2*K];              // State: u + i*v.  Registers.
	
    load_u_sym(u, v, u_in);               // Load data.  Macro defined in gmem.cu.
	for (int x = 0; x < L; x += L_ker)
    {
        int L_blk = (L_ker < L-x) ? L_ker : L-x;    // Layers in this block = min(L0, L-x)
        load_pos_cache_fwd;                         // Update cache, if necessary.
        load_T_sym;                                 // Load T matrices.

        for (int l = 0; l < L_blk; l++)             // Iterate through L_blk layers.
        {
            float temp1, temp2, temp3, u_2k, v_2k;
            if (shifts_cache[(x+l) % L_preload] % 2)        // MZIs not aligned with threads.  Warp shuffle.
            {
                for (int i = 0; i < K-1; i++)
                    matmult_sym(&T[l][3*i][threadIdx.x], u[2*i+1], v[2*i+1], u[2*i+2], v[2*i+2], temp1, temp2, temp3, true);
                u_2k = __shfl_down_sync(0xffffffffu, u[0], 1, 32); 
                v_2k = __shfl_down_sync(0xffffffffu, v[0], 1, 32);
                matmult_sym(&T[l][3*K-3][threadIdx.x], u[2*K-1], v[2*K-1], u_2k, v_2k, 
                            temp1, temp2, temp3, threadIdx.x != 31);
                u_2k = __shfl_up_sync(0xffffffffu, u_2k, 1, 32); 
                v_2k = __shfl_up_sync(0xffffffffu, v_2k, 1, 32);
                if (threadIdx.x) {u[0] = u_2k; v[0] = v_2k;}
            }
            else
                for (int i = 0; i < K; i++)                 // MZIs aligned with threads.  Easy case!
                    matmult_sym(&T[l][3*i][threadIdx.x], u[2*i], v[2*i], u[2*i+1], v[2*i+1], temp1, temp2, temp3, true);
        }
        
        p += L_ker * ldp;
        if (s) {s += L_ker * lds;}
        __syncthreads();
    }
    save_u_sym(u, v, u_out);               // Save data.
}
#endif

#undef L_ker
#undef L_preload
#undef K
#undef L0
#undef nL
#undef fname